#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "vector.h"
#include "config.h"

//Global values and accelerations
vector3* vals;
vector3** accels;

//Parallel implementation
__global__ void parallelCompute(vector3* vals, vector3** accels, vector3* d_vel, vector3* d_pos, double* d_mass){
}

//Memory allocation and driver code
void compute(){
}