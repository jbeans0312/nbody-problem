#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "vector.h"
#include "config.h"

//Global values and accelerations
vector3* vals;
vector3** accels;

//Parallel implementation
__global__ void parallelCompute(vector3* vals, vector3** accels, vector3* d_vel, vector3* d_pos, double* d_mass){
    int myId = blockIdx.x * blockDim.x + threadId.x
    int i = myId / NUMENTITIES;
    int j = myId % NUMENTITIES

    accels[myId] = &values[myId*NUMENTITIES];

    if(myId < NUMENTITIES * NUMENTITIES){
        if(i == j){
            FILL_VECTOR(accels[i][j],0,0,0);
        }else{
            vector3 distance;

            //calculate distance in 3D
            distance[0]=d_pos[i][0]-d_pos[j][0];
            distance[1]=d_pos[i][1]-d_pos[j][1];
            distance[2]=d_pos[i][2]-d_pos[j][2];

            //calculate acceleration values
            //fun fun fun physics calculation stuff
            double magnitude_sq=distance[0]*distance[0]+distance[1]*distance[1]+distance[2]*distance[2];
            double magnitude=sqrt(magnitude_sq);
			double accelmag=-1*GRAV_CONSTANT*d_mass[j]/magnitude_sq;
            FILL_VECTOR(accels[i][j],accelmag*distance[0]/magnitude,accelmag*distance[1]/magnitude,accelmag*distance[2]/magnitude);
        }

        vector3 accel_sum = {(double) *(accels[myId])[0], (double) *(accels[myId])[1], (double) *(accels[myId])[2]};

        d_vel[i][0]+=accel_sum[0]*INTERVAL;
		d_pos[i][0]=d_vel[i][0]*INTERVAL;

		d_vel[i][1]+=accel_sum[1]*INTERVAL;
		d_pos[i][1]=d_vel[i][1]*INTERVAL;

		d_vel[i][2]+=accel_sum[2]*INTERVAL;
		d_pos[i][2]=d_vel[i][2]*INTERVAL;
    }
}


//Memory allocation and driver code
void compute(){
}